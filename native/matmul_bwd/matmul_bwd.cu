#include "hip/hip_runtime.h"
#include "../common/accessor.cuh"
#include "../common/gpu_assert.cuh"
#include "../common/kernel_function.cuh"
#include "../common/utils.h"
#include "matmul_bwd.h"

#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void
kernel_matmul_cuda_kernel_bwd(const BatchLayout<KM_BATCH_DIM> batch_layout,
                              const BatchedAccessor<float, KM_BATCH_DIM, 1> x1_batch,
                              const BatchedAccessor<float, KM_BATCH_DIM, 1> x2_batch,
                              const BatchedAccessor<float, KM_BATCH_DIM, 2> rhs_batch,
                              const BatchedAccessor<float, KM_BATCH_DIM, 1> params_batch,
                              const BatchedAccessor<int, KM_BATCH_DIM, 1> start_batch,
                              const BatchedAccessor<int, KM_BATCH_DIM, 1> end_batch,
                              const BatchedAccessor<float, KM_BATCH_DIM, 2> out_grad_batch,
                              BatchedAccessor<float, KM_BATCH_DIM, 3> params_grad_batch) {
    // This is almost the same as the forward pass, as we have the same matmul structure in the
    // derivative. The main difference is that we can now accumulate the gradients to smaller
    // portions directly, saving registers. For this, we need the output gradient, which we will
    // load from global memory directly and cache in registers. However, we will need significantly
    // more shm and registers as we now need to buffer all gradients of the kernel function.

    // Load batch
    const auto batch = batch_layout.get_batch(blockIdx.z);
    const auto x1 = x1_batch[batch];
    const auto x2 = x2_batch[batch];
    const auto rhs = rhs_batch[batch];
    const auto params = params_batch[batch];
    const auto start = start_batch[batch];
    const auto end = end_batch[batch];
    const auto out_grad = out_grad_batch[batch];
    auto params_grad = params_grad_batch[batch];

    // Index calculations
    const int block_size = KM_BLOCK_SIZE;
    const int thread_dim = KM_MATMUL_BWD_THREAD_DIM;
    const int per_thread = KM_MATMUL_BWD_PER_THREAD;
    const int num_params = KM_NUM_PARAMS;
    static_assert(thread_dim * per_thread == block_size,
                  "block_size must be the product of thread_dim and per_thread");
    static_assert((thread_dim * thread_dim) % 32 == 0,
                  "Thread block must be evenly divisible in warps.");
    const int k_base = blockIdx.x * block_size;
    const int m_base = blockIdx.y * block_size;
    const int k_size = rhs.size(1);
    const int m_size = x1.size(0);

    // This is an alternative indexing that is used for loading from global to shared memory to
    // avoid bank conflicts.
    const auto thread_rank = threadIdx.y * thread_dim + threadIdx.x;
    const auto warp_based_x = thread_rank % 32;
    const auto warp_based_y = thread_rank / 32;
    const auto warp_num = thread_dim * thread_dim / 32;
    static_assert(thread_dim % warp_num == 0,
                  "thread_dim must be evenly divisible by the number of warps.");

    // Shared memory buffer
    // kernel_values: block_size, thread_dim
    // rhs: block_size, thread_dim
    extern __shared__ int sdata[];
    const int buffer_size = block_size * thread_dim;
    auto shm_rhs = (float *)sdata;
    auto shm_params_grad = shm_rhs + buffer_size;

    // Register buffer
    float reg_rhs[per_thread];
    float reg_params_grad[num_params][per_thread];
    float reg_out_grad[per_thread][per_thread];
#pragma unroll
    for (int m = 0; m < per_thread; m++) {
#pragma unroll
        for (int k = 0; k < per_thread; k++) {
            const auto m_index = m_base + m * thread_dim + threadIdx.y;
            const auto k_index = k_base + k * thread_dim + threadIdx.x;
            if (m_index < m_size && k_index < k_size) {
                reg_out_grad[m][k] = out_grad[m_index][k_index];
            } else {
                reg_out_grad[m][k] = 0;
            }
        }
    }

    // Load parameters to registers
    std::array<float, num_params> reg_params;
#pragma unroll
    for (int i = 0; i < num_params; i++) {
        reg_params[i] = params[i];
    }
    const int start_m = start[blockIdx.y];
    const int end_m = end[blockIdx.y];

    // Initialize accumulator for output
    // Each thread accumulates outputs for the entries
    // (m_base + m * thread_dim + threadIdx.y, k_base + k * thread_dim + threadIdx.x).
    // However, we only need a single accumulator per parameter now, as we can
    // multiply with the output gradient directly.
    float accumulator[num_params];
#pragma unroll
    for (int p = 0; p < num_params; p++) {
        accumulator[p] = 0;
    }

    // Outer loop advances blocks along columns of the kernel matrix and rows of the rhs
    for (int n_base = start_m; n_base < end_m; n_base += thread_dim) {
        // Load rhs and kernel matrix blocks into shared memory
        // n is always associated with threadIdx.y here to allow for coalesced access.
        // Trick: We can transpose the kernel matrix at virtually no cost here.
        // We use the warp-based indexing calculated above to avoid shm bank conflicts.
        for (int j = warp_based_y; j < thread_dim; j += warp_num) {
            for (int i = warp_based_x; i < block_size; i += 32) {
                const auto shm_index = j * block_size + i;
                const auto n = n_base + j;
                const auto k = k_base + i;
                const auto m = m_base + i;
                if (k < k_size && n < end_m) {
                    shm_rhs[shm_index] = rhs[n][k];
                } else {
                    shm_rhs[shm_index] = 0;
                }
                if (m < m_size && n < end_m) {
                    const auto grads = kernel_function_bwd(x1[m], x2[n], reg_params);
#pragma unroll
                    for (int p = 0; p < num_params; p++) {
                        shm_params_grad[p * buffer_size + shm_index] = grads[p];
                    }
                } else {
#pragma unroll
                    for (int p = 0; p < num_params; p++) {
                        shm_params_grad[p * buffer_size + shm_index] = 0;
                    }
                }
            }
        }
        __syncthreads();

        // Outer loop iterates over n.
        // We unroll all inner loops for ILP.
        for (int i = 0; i < thread_dim; i++) {
// Load from shm into registers
#pragma unroll
            for (int j = 0; j < per_thread; j++) {
                const auto shm_index = i * block_size + j * thread_dim;
                reg_rhs[j] = shm_rhs[shm_index + threadIdx.x];
#pragma unroll
                for (int p = 0; p < num_params; p++) {
                    reg_params_grad[p][j] =
                        shm_params_grad[p * buffer_size + shm_index + threadIdx.y];
                }
            }

// Inner loops iterate over m and k.
#pragma unroll
            for (int m = 0; m < per_thread; m++) {
#pragma unroll
                for (int k = 0; k < per_thread; k++) {
#pragma unroll
                    for (int p = 0; p < num_params; p++) {
                        accumulator[p] += reg_out_grad[m][k] * reg_params_grad[p][m] * reg_rhs[k];
                    }
                }
            }
        }
        __syncthreads();
    }

    // Write output to global memory
    const auto m = blockIdx.y * thread_dim + threadIdx.y;
    const auto k = blockIdx.x * thread_dim + threadIdx.x;
    if (m < params_grad.size(1) && k < params_grad.size(2)) {
#pragma unroll
        for (int p = 0; p < num_params; p++) {
            params_grad[p][m][k] = accumulator[p];
        }
    }
}

torch::Tensor kernel_matmul_bwd_cuda(torch::Tensor x1, torch::Tensor x2, torch::Tensor rhs,
                                     torch::Tensor params, torch::Tensor start, torch::Tensor end,
                                     torch::Tensor out_grad) {
    const int block_size = KM_BLOCK_SIZE;
    const int thread_dim = KM_MATMUL_BWD_THREAD_DIM;
    const int per_thread = KM_MATMUL_BWD_PER_THREAD;
    const int num_params = KM_NUM_PARAMS;
    const auto batch_layout = BatchLayout<KM_BATCH_DIM>(x1.sizes().data());

    const dim3 threads{thread_dim, thread_dim, 1};
    const dim3 blocks{KM_CEIL_DIV(rhs.size(-1), block_size), KM_CEIL_DIV(x1.size(-1), block_size),
                      batch_layout.num_batches()};
    const auto shared = (int)(rhs.element_size() * (1 + num_params) * block_size * thread_dim);

#ifdef KM_DEBUG_PRINT_SIZE
    printf("m, n, k: (%d, %d, %d, )\n", x1.size(-1), x2.size(-1), rhs.size(-1));
    printf("threads: (%d, %d, %d)\n", threads.x, threads.y, threads.z);
    printf("blocks: (%d, %d, %d)\n", blocks.x, blocks.y, blocks.z);
    printf("shared: %dK\n", KM_CEIL_DIV(shared, 1024));
#endif

    const auto out_opts =
        torch::TensorOptions().dtype(x1.dtype()).layout(x1.layout()).device(x1.device());
    const auto out_shape = batch_layout.make_shape<3>({num_params, x1.size(-1), rhs.size(-1)});
    auto params_grad = torch::zeros(out_shape, out_opts);

    const auto params_transformed = transform_params(params);

    kernel_matmul_cuda_kernel_bwd<<<blocks, threads, shared>>>(
        batch_layout, BatchedAccessor<float, KM_BATCH_DIM, 1>(x1),
        BatchedAccessor<float, KM_BATCH_DIM, 1>(x2), BatchedAccessor<float, KM_BATCH_DIM, 2>(rhs),
        BatchedAccessor<float, KM_BATCH_DIM, 1>(params_transformed),
        BatchedAccessor<int, KM_BATCH_DIM, 1>(start), BatchedAccessor<int, KM_BATCH_DIM, 1>(end),
        BatchedAccessor<float, KM_BATCH_DIM, 2>(out_grad),
        BatchedAccessor<float, KM_BATCH_DIM, 3>(params_grad));

    KM_DO_GPU_ASSERT;

    return transform_params_grad(params, params_grad.sum({-2, -1}));
}
