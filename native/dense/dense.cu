#include "hip/hip_runtime.h"
#include "../common/gpu_assert.cuh"
#include "../common/kernel_function.cuh"
#include "../common/utils.h"
#include "dense.h"

#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void kernel_dense_cuda_kernel(
    const torch::PackedTensorAccessor32<float, 1, torch::RestrictPtrTraits> x1,
    const torch::PackedTensorAccessor32<float, 1, torch::RestrictPtrTraits> x2,
    const torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> params,
    const torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> start,
    const torch::PackedTensorAccessor32<int, 1, torch::RestrictPtrTraits> end,
    torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> out) {

    const int block_size = KM_BLOCK_SIZE;
    const int thread_dim = KM_DENSE_THREAD_DIM;
    const int b = blockIdx.y;
    const int num_params = KM_NUM_PARAMS;
    static_assert(block_size % thread_dim == 0, "block_size must be divisible by thread_dim");
    const int per_thread = block_size / thread_dim;
    const int m_base = blockIdx.x * block_size;

    std::array<float, num_params> params_reg;
    for (int i = 0; i < num_params; i++) {
        params_reg[i] = params[i][b];
    }

    const int start_reg = start[blockIdx.x];
    const int end_reg = end[blockIdx.x];
    for (int n_base = start_reg; n_base < end_reg; n_base += block_size) {
        std::array<float, per_thread> x1_reg;
        for (int i = 0; i < per_thread; i++) {
            const int m = m_base + threadIdx.x + i * thread_dim;
            if (m < x1.size(0)) {
                x1_reg[i] = x1[m];
            } else {
                x1_reg[i] = 0;
            }
        }
        std::array<float, per_thread> x2_reg;
        for (int i = 0; i < per_thread; i++) {
            const int n = n_base + threadIdx.y + i * thread_dim;
            if (n < end_reg) {
                x2_reg[i] = x2[n];
            } else {
                x2_reg[i] = 0;
            }
        }
        for (int i = 0; i < per_thread; i++) {
            for (int j = 0; j < per_thread; j++) {
                const int m = m_base + threadIdx.x + i * thread_dim;
                const int n = n_base + threadIdx.y + j * thread_dim;
                if (m < x1.size(0) && n < end_reg) {
                    out[b][m][n] = kernel_function(x1_reg[i], x2_reg[j], params_reg);
                }
            }
        }
    }
}

torch::Tensor kernel_dense_cuda(torch::Tensor x1, torch::Tensor x2, torch::Tensor params,
                                torch::Tensor start, torch::Tensor end) {
    const int block_size = KM_BLOCK_SIZE;
    const int thread_dim = KM_DENSE_THREAD_DIM;
    const int b = params.size(1);

    const auto out_opts =
        torch::TensorOptions().dtype(x1.dtype()).layout(x1.layout()).device(x1.device());
    auto out = torch::zeros({b, x1.size(0), x2.size(0)}, out_opts);

    const dim3 blocks{KM_CEIL_DIV(x1.size(0), block_size), b, 1};
    const dim3 threads{thread_dim, thread_dim, 1};

    kernel_dense_cuda_kernel<<<blocks, threads>>>(
        x1.packed_accessor32<float, 1, torch::RestrictPtrTraits>(),
        x2.packed_accessor32<float, 1, torch::RestrictPtrTraits>(),
        params.packed_accessor32<float, 2, torch::RestrictPtrTraits>(),
        start.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
        end.packed_accessor32<int, 1, torch::RestrictPtrTraits>(),
        out.packed_accessor32<float, 3, torch::RestrictPtrTraits>());

    KM_DO_GPU_ASSERT;
    return out;
}
